#include <iostream>
#include <vector>
#include <fstream>
#include <hip/hip_runtime.h>
#include <sstream>
#include <iterator>
#include <hip/hip_runtime_api.h>
#include <cuda_device_runtime_api.h>

__global__ void matmul_tiled(const int N, const float *A, const float *B, float *C, int tileSizes[3]) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k_tile = blockIdx.z * blockDim.z + threadIdx.z;

    if (i < N && j < N && k_tile < N) {
        float sum = 0.0f;
        for (int k = k_tile; k < min(k_tile + tileSizes[2], N); k++) {
            sum += A[i * N + k] * B[k * N + j];
        }
        atomicAdd(&C[i * N + j], sum);
    }
}

void read_params(const std::string &filename, std::vector<std::vector<int>> &params) {
    std::ifstream file(filename);
    std::string line;
    while (std::getline(file, line)) {
        std::istringstream iss(line);
        std::vector<int> param_set((std::istream_iterator<int>(iss)), std::istream_iterator<int>());
        if (!param_set.empty()) {
            params.push_back(param_set);
        }
    }
}

int main() {
	// cudaProfilerStart();
    hipDeviceSynchronize();

    std::vector<std::vector<int>> params;
    read_params("params.txt", params);

    for (const auto &param_set : params) {
        int N = param_set[0];
        int tileSizes[3] = {param_set[1], param_set[2], param_set[3]};

        float *A, *B, *C;
        hipMallocManaged(&A, N * N * sizeof(float));
        hipMallocManaged(&B, N * N * sizeof(float));
        hipMallocManaged(&C, N * N * sizeof(float));

		std::cout << N << " " << tileSizes[0] << " " << tileSizes[1] << " " << tileSizes[2] << std::endl;

        for (int i = 0; i < N * N; i++) {
            A[i] = rand() % 10 + 1;
            B[i] = rand() % 10 + 1;
            C[i] = 0;
        }

        dim3 blockDim(tileSizes[0], tileSizes[1], tileSizes[2]);
        dim3 gridDim((N + blockDim.x - 1) / blockDim.x, (N + blockDim.y - 1) / blockDim.y, (N + blockDim.z - 1) / blockDim.z);
        matmul_tiled<<<gridDim, blockDim>>>(N, A, B, C, tileSizes);

        hipDeviceSynchronize();

        // Print the result matrix C (optional)
        // for (int i = 0; i < N; i++) {
        //     for (int j = 0; j < N; j++) {
        //         std::cout << C[i * N + j] << " ";
        //     }
        //     std::cout << std::endl;
        // }

        hipFree(A);
        hipFree(B);
        hipFree(C);
    }

  	// cudaProfilerStop();

	hipDeviceSynchronize();
    return 0;
}

